#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"
#include "gpu.cu"


int main() {
    int n_tests = 5;
    double *cpu_time = new double[n_tests];
    double *cpu_alloc_time = new double[n_tests];
    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    for (int t = 0; t < n_tests; t++) {
        u64 N = 1 << 28;
        u32 scale = 1 << 10;
        double start, end;

#pragma region CPU
        start = getSecond();
        u16 *cpu_avg = (u16 *) malloc(scale * sizeof(u16));
        u16 *cpu_min = (u16 *) malloc(scale * sizeof(u16));
        u16 *cpu_max = (u16 *) malloc(scale * sizeof(u16));
        u16 *cpu_back = (u16 *) malloc(scale * sizeof(u16));
        end = getSecond();
        cpu_alloc_time[t] = end - start;

        start = getSecond();
        simple_cpu_scaled(cpu_avg, cpu_min, cpu_max, N, scale);
        end = getSecond();
        cpu_time[t] = end - start;
#pragma endregion

#pragma region GPU
        warmup();

        start = getSecond();
        u16 *gpu_res1;
        hipMalloc(&gpu_res1, N * sizeof(u16));
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        int block_size = 1024;
        int grid_size = (N + block_size - 1) / block_size;
//        std::cout << "Grid size: " << grid_size << std::endl;
//        std::cout << "Block size: " << block_size << std::endl;
        simple_gpu<<<grid_size, block_size>>>(gpu_res1, N);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COPY BACK
        start = getSecond();
        hipMemcpy(res2, gpu_res1, N * sizeof(u16), hipMemcpyDeviceToHost);
        res2[0] = 0;
        end = getSecond();
        gpu_copy_time[t] = end - start;
#pragma endregion

        // COMPARISON
        compare_arrays(res1, res2, N);

        // FREE memory
        free(res1);
        free(res2);
        hipFree(gpu_res1);

        std::cout << "Test " << t << " completed" << std::endl;
    }

    std::cout << "CPU time: " << average(cpu_time, n_tests) << " s" << std::endl;
    std::cout << "CPU alloc time: " << average(cpu_alloc_time, n_tests) << " s" << std::endl;
    std::cout << "GPU time: " << average(gpu_time, n_tests) << " s" << std::endl;
    std::cout << "GPU alloc time: " << average(gpu_alloc_time, n_tests) << " s" << std::endl;
    std::cout << "GPU copy time: " << average(gpu_copy_time, n_tests) << " s" << std::endl;

    return 0;
}