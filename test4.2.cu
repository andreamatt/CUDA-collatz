#include "hip/hip_runtime.h"
#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"

#define BITS 9
#define TABLE_SIZE 512
#define BATCH_SIZE 1024

__device__ __constant__ u32 table_B[TABLE_SIZE];
__device__ __constant__ u32 table_C[TABLE_SIZE];
__device__ __constant__ u16 table_D[TABLE_SIZE];
__device__ __constant__ u16 table_E[TABLE_SIZE];


__global__ void gpu_LUT(u16 *res, u64 offset, u64 n_batches) {
    u64 id = blockIdx.x * blockDim.x + threadIdx.x;
    u64 i_start = BATCH_SIZE * id + offset;
    u64 i_end = i_start + BATCH_SIZE;
    u16 min_c = UINT16_MAX;
    u16 max_c = 0;
    u32 sum_c = 0;
    if (i_start == 0) i_start = 1;
    for (u64 i = i_start; i < i_end; i++) {
        u64 num = i;
        u16 count = 0;

        while (num >= TABLE_SIZE) {
            u64 n_high = num >> BITS;
            u64 n_low = num - (n_high << BITS);
            u32 b = table_B[n_low];
            u32 c = table_C[n_low];
            count += table_D[n_low];
            num = n_high * b + c;
        }
        // end of cycle, use LUT for remaining
        count += table_E[num];

        // update stats
        if (count < min_c) {
            min_c = count;
        }
        if (count > max_c) {
            max_c = count;
        }
        sum_c += count;
    }
    res[id] = sum_c / BATCH_SIZE;
    res[id + n_batches] = min_c;
    res[id + n_batches * 2] = max_c;
}

void generate_LUT(u32 *B_table, u32 *C_table, u16 *D_table) {
    u32 A = TABLE_SIZE;
    for (u32 i = A; i < 2 * A; i++) {
        u32 n_h = i >> BITS;
        u32 n_l = i - (n_h << BITS);
        u32 b = A;
        u32 c = n_l;
        u16 d = 0;
        while (true) {
            if (b % 2 == 0) {
                if (c % 2 == 0) {
                    b = b / 2;
                    c = c / 2;
                } else {
                    b = b * 3;
                    c = c * 3 + 1;
                }
                d++;
            } else {
                B_table[n_l] = b;
                C_table[n_l] = c;
                D_table[n_l] = d;
                break;
            }
        }
    }
}

int main() {
    bool verify = false;
    int n_tests = 1;
    u64 N_to_calc = power(30);
    u64 offset = power(40);
    u64 n_batches = N_to_calc / BATCH_SIZE;
    u64 n_threads = n_batches;
    u64 block_size = 1024;
    u64 grid_size = n_threads / block_size;

    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    // defines the three tables
    u32 *table_B_cpu = (u32 *) malloc(TABLE_SIZE * sizeof(u32));
    u32 *table_C_cpu = (u32 *) malloc(TABLE_SIZE * sizeof(u32));
    u16 *table_D_cpu = (u16 *) malloc(TABLE_SIZE * sizeof(u16));
    u16 *table_E_cpu = (u16 *) malloc(TABLE_SIZE * sizeof(u16));
    // calculate the three tables
    std::cout << "CPU started" << std::endl;
    generate_LUT(table_B_cpu, table_C_cpu, table_D_cpu);
    std::cout << "LUT generation finished" << std::endl;
    dynamic_cpu(table_E_cpu, TABLE_SIZE);
    std::cout << "Table E finished" << std::endl;

    // calculate result for cpu
    std::cout << "CPU started" << std::endl;
    u16 *cpu_res = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    u16 *cpu_res_compare = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    if (verify) {
        simple_cpu_batch(cpu_res, offset, n_batches, BATCH_SIZE);
    }
    std::cout << "CPU finished" << std::endl;

    warmup();

    for (int t = 0; t < n_tests; t++) {
        double start, end;

        start = getSecond();
        // copy all 4 tables to gpu constant memory
        hipMemcpyToSymbol(HIP_SYMBOL(table_B), table_B_cpu, TABLE_SIZE * sizeof(u32), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(table_C), table_C_cpu, TABLE_SIZE * sizeof(u32), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(table_D), table_D_cpu, TABLE_SIZE * sizeof(u16), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(table_E), table_E_cpu, TABLE_SIZE * sizeof(u16), 0, hipMemcpyHostToDevice);
        // allocate gpu memory for result
        u16 *gpu_res;
        hipMalloc(&gpu_res, n_batches * 3 * sizeof(u16));
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        gpu_LUT<<<grid_size, block_size>>>(gpu_res, offset, n_batches);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COMPARISON
        bool success = true;
        if (verify) {
            // COPY BACK
            start = getSecond();
            hipMemcpy(cpu_res_compare, gpu_res, n_batches * 3 * sizeof(u16), hipMemcpyDeviceToHost);
            end = getSecond();
            gpu_copy_time[t] = end - start;
            success = compare_arrays(cpu_res, cpu_res_compare, n_batches * 3);
        } else {
            gpu_copy_time[t] = 0;
        }

        // FREE memory
        hipFree(gpu_res);

        if (!success) {
            std::cout << "Test " << t << " failed" << std::endl;
            return 1;
        }
        std::cout << "Test " << t << " completed" << std::endl;
    }

    return print_stats_gpu(gpu_time, gpu_alloc_time, gpu_copy_time, n_tests);
}