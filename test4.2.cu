#include "hip/hip_runtime.h"
#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"

#define BITS 12
#define TABLE_SIZE 4096
#define BATCH_SIZE 1024

__global__ void
gpu_LUT(u16 *res, u32 *gmem_table_B, u32 *gmem_table_C, u16 *gmem_table_D, u16 *gmem_table_E, u64 offset,
        u64 n_batches) {
    __shared__ u32 table_B[TABLE_SIZE];
    __shared__ u32 table_C[TABLE_SIZE];
    __shared__ u16 table_D[TABLE_SIZE];
    __shared__ u16 table_E[TABLE_SIZE];

    if (TABLE_SIZE > BATCH_SIZE) {
        int iters = TABLE_SIZE / BATCH_SIZE;
        for (int i = 0; i < iters; i++) {
            int idx = i * BATCH_SIZE + threadIdx.x;
            if (idx < TABLE_SIZE) {
                table_B[idx] = gmem_table_B[idx];
                table_C[idx] = gmem_table_C[idx];
                table_D[idx] = gmem_table_D[idx];
                table_E[idx] = gmem_table_E[idx];
            }
        }
    } else if (TABLE_SIZE == BATCH_SIZE) {
        table_B[threadIdx.x] = gmem_table_B[threadIdx.x];
        table_C[threadIdx.x] = gmem_table_C[threadIdx.x];
        table_D[threadIdx.x] = gmem_table_D[threadIdx.x];
        table_E[threadIdx.x] = gmem_table_E[threadIdx.x];
    } else {
        int idx = threadIdx.x;
        if (idx < TABLE_SIZE) {
            table_B[idx] = gmem_table_B[idx];
            table_C[idx] = gmem_table_C[idx];
            table_D[idx] = gmem_table_D[idx];
            table_E[idx] = gmem_table_E[idx];
        }
    }
    __syncthreads();

    u64 id = blockIdx.x * blockDim.x + threadIdx.x;
    u64 i_start = BATCH_SIZE * id + offset;
    u64 i_end = i_start + BATCH_SIZE;
    u16 min_c = UINT16_MAX;
    u16 max_c = 0;
    u32 sum_c = 0;
    if (i_start == 0) i_start = 1;
    for (u64 i = i_start; i < i_end; i++) {
        u64 num = i;
        u16 count = 0;

        while (num >= TABLE_SIZE) {
            u64 n_high = num >> BITS;
            u64 n_low = num - (n_high << BITS);
            u32 b = table_B[n_low];
            u32 c = table_C[n_low];
            count += table_D[n_low];
            num = n_high * b + c;
        }
        // end of cycle, use LUT for remaining
        count += table_E[num];

        // update stats
        if (count < min_c) {
            min_c = count;
        }
        if (count > max_c) {
            max_c = count;
        }
        sum_c += count;
    }
    res[id] = sum_c / BATCH_SIZE;
    res[id + n_batches] = min_c;
    res[id + n_batches * 2] = max_c;
}

void generate_LUT(u32 *B_table, u32 *C_table, u16 *D_table) {
    u32 A = TABLE_SIZE;
    for (u32 i = A; i < 2 * A; i++) {
        u32 n_h = i >> BITS;
        u32 n_l = i - (n_h << BITS);
        u32 b = A;
        u32 c = n_l;
        u16 d = 0;
        while (true) {
            if (b % 2 == 0) {
                if (c % 2 == 0) {
                    b = b / 2;
                    c = c / 2;
                } else {
                    b = b * 3;
                    c = c * 3 + 1;
                }
                d++;
            } else {
                B_table[n_l] = b;
                C_table[n_l] = c;
                D_table[n_l] = d;
                break;
            }
        }
    }
}

int main() {
    bool verify = false;
    int n_tests = 10;
    u64 N_to_calc = power(34);
    u64 offset = power(40);
    u64 n_batches = N_to_calc / BATCH_SIZE;
    u64 n_threads = n_batches;
    u64 block_size = BATCH_SIZE;
    u64 grid_size = n_threads / block_size;

    // print parameters
    std::cout << "N_to_calc: " << N_to_calc << std::endl;
    std::cout << "BITS: " << BITS << " TABLE_SIZE: " << TABLE_SIZE << std::endl;
    std::cout << "n_batches: " << n_batches << " n_threads: " << n_threads << " block_size: " << block_size
              << " grid_size: " << grid_size << std::endl;

    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    // defines the three tables
    u32 *table_B_cpu = (u32 *) malloc(TABLE_SIZE * sizeof(u32));
    u32 *table_C_cpu = (u32 *) malloc(TABLE_SIZE * sizeof(u32));
    u16 *table_D_cpu = (u16 *) malloc(TABLE_SIZE * sizeof(u16));
    u16 *table_E_cpu = (u16 *) malloc(TABLE_SIZE * sizeof(u16));
    // calculate the three tables
    std::cout << "CPU started" << std::endl;
    generate_LUT(table_B_cpu, table_C_cpu, table_D_cpu);
    std::cout << "LUT generation finished" << std::endl;
    dynamic_cpu(table_E_cpu, TABLE_SIZE);
    std::cout << "Table E finished" << std::endl;

    // calculate result for cpu
    std::cout << "CPU started" << std::endl;
    u16 *cpu_res = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    u16 *cpu_res_compare = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    if (verify) {
        simple_cpu_batch(cpu_res, offset, n_batches, BATCH_SIZE);
    }
    std::cout << "CPU finished" << std::endl;

    // configure shared memory
    auto error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
    if (error != hipSuccess) {
        std::cout << "Error setting shared memory configuration" << std::endl;
        return 1;
    }

    warmup();

    for (int t = 0; t < n_tests; t++) {
        double start, end;

        start = getSecond();
        // copy all 4 tables to gpu constant memory
        u32 *table_B_gpu;
        u32 *table_C_gpu;
        u16 *table_D_gpu;
        u16 *table_E_gpu;
        hipMalloc((void **) &table_B_gpu, TABLE_SIZE * sizeof(u32));
        hipMalloc((void **) &table_C_gpu, TABLE_SIZE * sizeof(u32));
        hipMalloc((void **) &table_D_gpu, TABLE_SIZE * sizeof(u16));
        hipMalloc((void **) &table_E_gpu, TABLE_SIZE * sizeof(u16));
        hipMemcpy(table_B_gpu, table_B_cpu, TABLE_SIZE * sizeof(u32), hipMemcpyHostToDevice);
        hipMemcpy(table_C_gpu, table_C_cpu, TABLE_SIZE * sizeof(u32), hipMemcpyHostToDevice);
        hipMemcpy(table_D_gpu, table_D_cpu, TABLE_SIZE * sizeof(u16), hipMemcpyHostToDevice);
        hipMemcpy(table_E_gpu, table_E_cpu, TABLE_SIZE * sizeof(u16), hipMemcpyHostToDevice);
        // allocate gpu memory for result
        u16 *gpu_res;
        hipMalloc(&gpu_res, n_batches * 3 * sizeof(u16));
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        gpu_LUT<<<grid_size, block_size>>>(gpu_res, table_B_gpu, table_C_gpu, table_D_gpu, table_E_gpu, offset,
                                           n_batches);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COMPARISON
        bool success = true;
        if (verify) {
            // COPY BACK
            start = getSecond();
            hipMemcpy(cpu_res_compare, gpu_res, n_batches * 3 * sizeof(u16), hipMemcpyDeviceToHost);
            end = getSecond();
            gpu_copy_time[t] = end - start;
            success = compare_arrays(cpu_res, cpu_res_compare, n_batches * 3);
        } else {
            gpu_copy_time[t] = 0;
        }

        // FREE memory
        hipFree(gpu_res);
        hipFree(table_B_gpu);
        hipFree(table_C_gpu);
        hipFree(table_D_gpu);
        hipFree(table_E_gpu);

        if (!success) {
            std::cout << "Test " << t << " failed" << std::endl;
            return 1;
        }
        std::cout << "Test " << t << " completed" << std::endl;
    }

    return print_stats_gpu(gpu_time, gpu_alloc_time, gpu_copy_time, n_tests);
}