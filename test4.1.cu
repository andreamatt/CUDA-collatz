#include "hip/hip_runtime.h"
#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"

#define BITS 14
#define TABLE_SIZE 16384
#define BATCH_SIZE 1024

__global__ void gpu_LUT_E(u16 *res, u16 *gmem_table_E, u64 offset, u64 n_batches) {
    __shared__ u16 table_E[TABLE_SIZE];

    if (TABLE_SIZE > BATCH_SIZE) {
        int iters = TABLE_SIZE / BATCH_SIZE;
        for (int i = 0; i < iters; i++) {
            int idx = i * BATCH_SIZE + threadIdx.x;
            if (idx < TABLE_SIZE) {
                table_E[idx] = gmem_table_E[idx];
            }
        }
    } else if (TABLE_SIZE == BATCH_SIZE) {
        table_E[threadIdx.x] = gmem_table_E[threadIdx.x];
    } else {
        int idx = threadIdx.x;
        if (idx < TABLE_SIZE) {
            table_E[idx] = gmem_table_E[idx];
        }
    }
    __syncthreads();

    u64 id = blockIdx.x * blockDim.x + threadIdx.x;
    u64 i_start = BATCH_SIZE * id + offset;
    u64 i_end = i_start + BATCH_SIZE;
    u16 min_c = UINT16_MAX;
    u16 max_c = 0;
    u32 sum_c = 0;
    if (i_start == 0) i_start = 1;
    for (u64 i = i_start; i < i_end; i++) {
        u64 a = i;
        u16 c = 0;
        while (a >= TABLE_SIZE) {
            if (a % 2 == 0) {
                a = a / 2;
                c++;
            } else {
                a = (3 * a + 1) / 2;
                c += 2;
            }
        }
        // LUT with table_E
        c += table_E[a];

        if (c < min_c) {
            min_c = c;
        }
        if (c > max_c) {
            max_c = c;
        }
        sum_c += c;
    }
    res[id] = sum_c / BATCH_SIZE;
    res[id + n_batches] = min_c;
    res[id + n_batches * 2] = max_c;
}

int main() {
    bool verify = false;
    int n_tests = 10;
    u64 N_to_calc = power(34);
    u64 offset = power(40);
    u64 n_batches = N_to_calc / BATCH_SIZE;
    u64 n_threads = n_batches;
    u64 block_size = BATCH_SIZE;
    u64 grid_size = n_threads / block_size;

    // print parameters
    std::cout << "N_to_calc: " << N_to_calc << std::endl;
    std::cout << "BITS: " << BITS << " TABLE_SIZE: " << TABLE_SIZE << std::endl;
    std::cout << "n_batches: " << n_batches << " n_threads: " << n_threads << " block_size: " << block_size
              << " grid_size: " << grid_size << std::endl;

    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    // defines the three tables
    u16 *table_E_cpu = (u16 *) malloc(TABLE_SIZE * sizeof(u16));
    // calculate the three tables
    std::cout << "CPU started" << std::endl;
    dynamic_cpu(table_E_cpu, TABLE_SIZE);
    std::cout << "Table E finished" << std::endl;

    // calculate result for cpu
    std::cout << "CPU started" << std::endl;
    u16 *cpu_res = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    u16 *cpu_res_compare = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    if (verify) {
        simple_cpu_batch(cpu_res, offset, n_batches, BATCH_SIZE);
    }
    std::cout << "CPU finished" << std::endl;

    warmup();

    for (int t = 0; t < n_tests; t++) {
        double start, end;

        start = getSecond();
        // copy table to gpu memory
        u16 *gmem_table_E;
        hipMalloc(&gmem_table_E, TABLE_SIZE * sizeof(u16));
        hipMemcpy(gmem_table_E, table_E_cpu, TABLE_SIZE * sizeof(u16), hipMemcpyHostToDevice);
        // allocate gpu memory for result
        u16 *gpu_res;
        hipMalloc(&gpu_res, n_batches * 3 * sizeof(u16));
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        gpu_LUT_E<<<grid_size, block_size>>>(gpu_res, gmem_table_E, offset, n_batches);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COMPARISON
        bool success = true;
        if (verify) {
            // COPY BACK
            start = getSecond();
            hipMemcpy(cpu_res_compare, gpu_res, n_batches * 3 * sizeof(u16), hipMemcpyDeviceToHost);
            end = getSecond();
            gpu_copy_time[t] = end - start;
            success = compare_arrays(cpu_res, cpu_res_compare, n_batches * 3);
        } else {
            gpu_copy_time[t] = 0;
        }

        // FREE memory
        hipFree(gpu_res);
        hipFree(gmem_table_E);

        if (!success) {
            std::cout << "Test " << t << " failed" << std::endl;
            return 1;
        }
        std::cout << "Test " << t << " completed" << std::endl;
    }

    return print_stats_gpu(gpu_time, gpu_alloc_time, gpu_copy_time, n_tests);
}