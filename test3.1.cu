#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"
#include "gpu.cu"


int main() {
    int n_tests = 1;
    double *cpu_time = new double[n_tests];
    double *cpu_alloc_time = new double[n_tests];
    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    for (int t = 0; t < n_tests; t++) {
        u64 N = 1 << 30;
        u32 batch_size = 1 << 10;
        u64 N_batches = N / batch_size;
        u64 arr_size = 3 * N_batches;
        double start, end;

#pragma region CPU
        start = getSecond();
        u16 *cpu_res1 = (u16 *) malloc(arr_size * sizeof(u16));
        u16 *cpu_res2 = (u16 *) malloc(arr_size * sizeof(u16));
        end = getSecond();
        cpu_alloc_time[t] = end - start;

        start = getSecond();
        simple_cpu_batch(cpu_res1, N, N_batches, batch_size);
        end = getSecond();
        cpu_time[t] = end - start;
#pragma endregion

#pragma region GPU
        warmup();

        start = getSecond();
        u16 *gpu_res;
        hipMalloc(&gpu_res, arr_size * sizeof(u16));
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        int block_size = 1024;
        int grid_size = (N + block_size - 1) / (block_size * batch_size);
//        std::cout << "Grid size: " << grid_size << std::endl;
//        std::cout << "Block size: " << block_size << std::endl;
        simple_gpu_batch_1<<<grid_size, block_size>>>(gpu_res, N, N_batches, batch_size);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COPY BACK
        start = getSecond();
        hipMemcpy(cpu_res2, gpu_res, arr_size * sizeof(u16), hipMemcpyDeviceToHost);
        end = getSecond();
        gpu_copy_time[t] = end - start;
#pragma endregion

        // COMPARISON
//        bool success = compare_arrays(cpu_res1, cpu_res2, arr_size);

        // FREE memory
        free(cpu_res1);
        free(cpu_res2);
        hipFree(gpu_res);

//        if (!success) {
//            std::cout << "Test " << t << " failed" << std::endl;
//            return 1;
//        }
//        std::cout << "Test " << t << " completed" << std::endl;
    }

    return print_stats(cpu_time, cpu_alloc_time, gpu_time, gpu_alloc_time, gpu_copy_time, n_tests);
}