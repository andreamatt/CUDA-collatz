#include "hip/hip_runtime.h"
#include <iostream>
#include "cpu.cu"
#include "utils/time.cu"
#include "utils/warmup.cu"
#include "utils/stats.cu"

#define BATCH_SIZE 1024
#define MEMORY_POWER 31

__global__ void simple_gpu(u16 *res) {
    u64 i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0) {
        u64 a = i;
        u16 c = 0;
        while (a != 1) {
            if (a % 2 == 0) {
                a = a / 2;
                c++;
            } else {
                a = (3 * a + 1) / 2;
                c += 2;
            }
        }
        res[i] = c;
    } else {
        res[i] = 0;
    }
}

__global__ void gpu_LUT(u16 *res, u64 offset, u64 n_batches, u16 *table_E, u64 table_size) {
    u64 id = blockIdx.x * blockDim.x + threadIdx.x;
    u64 i_start = BATCH_SIZE * id + offset;
    u64 i_end = i_start + BATCH_SIZE;
    u16 min_c = UINT16_MAX;
    u16 max_c = 0;
    u32 sum_c = 0;
    if (i_start == 0) i_start = 1;
    for (u64 i = i_start; i < i_end; i++) {
        u64 a = i;
        u16 count = 0;
        while (a >= table_size) {
            if (a % 2 == 0) {
                a = a / 2;
                count++;
            } else {
                a = (3 * a + 1) / 2;
                count += 2;
            }
        }
        // LUT with table_E
        count += table_E[a];

        // update stats
        if (count < min_c) {
            min_c = count;
        }
        if (count > max_c) {
            max_c = count;
        }
        sum_c += count;
    }
    res[id] = sum_c / BATCH_SIZE;
    res[id + n_batches] = min_c;
    res[id + n_batches * 2] = max_c;
}

int main() {
    bool verify = false;
    int n_tests = 10;
    u64 table_size = power(MEMORY_POWER);
    u64 N_to_calc = power(32);
    u64 offset = power(40);
    u64 n_batches = N_to_calc / BATCH_SIZE;
    u64 n_threads = n_batches;
    u64 block_size = BATCH_SIZE;
    u64 grid_size = n_threads / block_size;

    // print parameters
    std::cout << "N_to_calc: " << N_to_calc << std::endl;
    std::cout << "BITS: " << BITS << " TABLE_SIZE: " << TABLE_SIZE << std::endl;
    std::cout << "n_batches: " << n_batches << " n_threads: " << n_threads << " block_size: " << block_size
              << " grid_size: " << grid_size << std::endl;

    double *gpu_time = new double[n_tests];
    double *gpu_alloc_time = new double[n_tests];
    double *gpu_copy_time = new double[n_tests];

    // calculate result for cpu
    std::cout << "CPU started" << std::endl;
    u16 *cpu_res = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    u16 *cpu_res_compare = (u16 *) malloc(n_batches * 3 * sizeof(u16));
    if (verify) {
        simple_cpu_batch(cpu_res, offset, n_batches, BATCH_SIZE);
    }
    std::cout << "CPU finished" << std::endl;

    warmup();

    for (int t = 0; t < n_tests; t++) {
        double start, end;

        start = getSecond();
        // allocate gpu memory for table E
        u16 *table_E_gpu;
        hipMalloc(&table_E_gpu, table_size * sizeof(u16));
        // allocate gpu memory for result
        u16 *gpu_res;
        hipMalloc(&gpu_res, n_batches * 3 * sizeof(u16));
        // calculate table E
        simple_gpu<<<table_size / block_size, block_size>>>(table_E_gpu);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_alloc_time[t] = end - start;

        // GPU work
        start = getSecond();
        gpu_LUT<<<grid_size, block_size>>>(gpu_res, offset, n_batches, table_E_gpu, table_size);
        hipDeviceSynchronize();
        end = getSecond();
        gpu_time[t] = end - start;


        // CHECK CUDA ERRORS
        auto code = hipGetLastError();
        if (code != hipSuccess) {
            std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        }

        // COMPARISON
        bool success = true;
        if (verify) {
            // COPY BACK
            start = getSecond();
            hipMemcpy(cpu_res_compare, gpu_res, n_batches * 3 * sizeof(u16), hipMemcpyDeviceToHost);
            end = getSecond();
            gpu_copy_time[t] = end - start;
            success = compare_arrays(cpu_res, cpu_res_compare, n_batches * 3);
        } else {
            gpu_copy_time[t] = 0;
        }

        // FREE memory
        hipFree(gpu_res);
        hipFree(table_E_gpu);

        if (!success) {
            std::cout << "Test " << t << " failed" << std::endl;
            return 1;
        }
        std::cout << "Test " << t << " completed" << std::endl;
    }

    return print_stats_gpu(gpu_time, gpu_alloc_time, gpu_copy_time, n_tests);
}