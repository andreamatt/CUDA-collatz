
#include <hip/hip_runtime.h>
double average(double *data, int n) {
    double sum = 0;
    for (int i = 0; i < n; i++) {
        sum += data[i];
    }
    return sum / n;
}

double max(double *data, int n) {
    double max = data[0];
    for (int i = 0; i < n; i++) {
        if (data[i] > max) {
            max = data[i];
        }
    }
    return max;
}

double min(double *data, int n) {
    double min = data[0];
    for (int i = 0; i < n; i++) {
        if (data[i] < min) {
            min = data[i];
        }
    }
    return min;
}